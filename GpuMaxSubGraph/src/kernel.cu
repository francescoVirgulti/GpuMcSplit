#include "hip/hip_runtime.h"
//
// Created by davide on 5/3/24.
//

#include "gpu_header.hpp"

#include <vector>
#include <string.h>
#include <string>
#include <queue>
#include <algorithm>
#include <unordered_set>
using namespace std;


//struct 
typedef struct{
    int g_size;
    int h_size;
    int row_ring_size;
    int *col_ring_size;
    int *g;
    int *h;
    int adj;
    char label[4];
    int **rings_g;
}GpuLabelClass;


typedef struct{
    int first;
    int second;
}Pair;

Pair *m_best_solution;


typedef struct {
    int labels_size;
    int m_size;
    GpuLabelClass *labels;
    GpuLabelClass single_label;
    int *idxList;
    Pair *m_local;
}ThreadVar;

//auto is for autonomous

ThreadVar **thread_pool_list;


int *auto_pool_size;
Pair **auto_pool_m_best;
int *auto_pool_len_m_best;
ThreadVar *auto_pool_tmp;

vector<int> length_list;



float *gpu_edge_labels;
int size_edge_labels;


float **gpu_g0;
int size_gpu_g0_row;
int size_gpu_g0_col;

float **gpu_g1;
int size_gpu_g1_row;
int size_gpu_g1_col;








void printLabelClass(GpuLabelClass lb) {
        cout<< lb.label << " [ ";
        cout<< " G("<< lb.g_size << "): ";
        if(!lb.g_size == 0) {for ( int i = 0 ; i < lb.g_size; i++ ) cout<<"["<<lb.g[i]<<"]";}
        cout<< " H("<< lb.h_size << "): ";
        if(!lb.h_size == 0 ) {for ( int i = 0; i <  lb.h_size; i++ ) cout<<"["<<lb.h[i]<<"]";}
        cout<< " RINGS("<< lb.row_ring_size << "): [";
        for( int i = 0; i< lb.row_ring_size; i++){cout<<"("<<lb.col_ring_size[i]<<")"<<"["; for( int j = 0; j <  lb.col_ring_size[i]; j++) cout<<lb.rings_g[i][j]<<", ";  cout<<" ]";}
        cout<<"]";
        cout<< " edge : " <<lb.adj<<" " ;
        cout<< lb.label << " ] "<<endl;
}   
void copyIntArray(int *a, int *b, int sizeb){
    for ( int i = 0 ; i < sizeb ; i++){
        a[i] = b[i];
    }
}
void copyIntMatrix(int **a, int **b, int rowsize, int *colsize )
{

    for( int i = 0 ; i < rowsize ; i++){
        for( int j = 0 ; j < colsize[i] ; j++){
            a[i][j] = b[i][j];
        }
    }

}
void cpyGpuLabelClass(GpuLabelClass *l1, GpuLabelClass l2){
    l1->adj = l2.adj;
    l1->row_ring_size = l2.row_ring_size;
    l1->g_size = l2.g_size;
    l1->h_size = l2.h_size;
    strcpy(l1->label , l2.label);
    copyIntArray( l1->g , l2.g, l2.g_size);
    copyIntArray( l1->h, l2.h, l2.h_size);
    copyIntArray( l1->col_ring_size, l2.col_ring_size , l2.row_ring_size);
    copyIntMatrix( l1->rings_g, l2.rings_g, l2.row_ring_size, l2.col_ring_size );/**/
}
void vectorToPointerEdge(float *gpu_edge_labels){
    if(edge_labels.size() == 0){
        gpu_edge_labels = nullptr;
        return;
    }
    int size = 0;
    for(float edg : edge_labels){
        gpu_edge_labels[size] = edg;
        size++;
    }

    return;
}
void vectorToPointerMatrix(const std::vector<std::vector<float>>& g,float** gpu_g) {
    // Get dimensions of the vector
    int numRows = g.size();
    if (numRows == 0) {
        // Empty vector, set pointers to nullptr
        gpu_g = nullptr;
        return;
    }

    int numCol = g[0].size();

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCol; ++j) {
            gpu_g[i][j] = g[i][j];
        }
    }
}
void LabelFromCpuToGpu(GpuLabelClass *new_label, const vector<LabelClass>& old_label ){

    for (int idx = 0 ; idx < old_label.size() ; ++idx ){
        new_label[idx].g_size = old_label.at(idx).g.size();
        new_label[idx].h_size = old_label.at(idx).h.size();
        new_label[idx].row_ring_size = old_label.at(idx).rings_g.size();
        new_label[idx].adj = old_label.at(idx).adj;
        strcpy(new_label[idx].label, old_label.at(idx).label.c_str() );

        for ( int j = 0 ; j < old_label.at(idx).g.size() ; ++j ){
            new_label[idx].g[j] = old_label.at(idx).g.at(j);
        }
        for ( int j = 0 ; j < old_label.at(idx).h.size() ; ++j ){
            new_label[idx].h[j] = old_label.at(idx).h.at(j);}

        for ( int row = 0 ; row < old_label.at(idx).rings_g.size() ; ++row ){
            new_label[idx].col_ring_size[row] = old_label.at(idx).rings_g.at(row).size();
            for ( int col = 0 ; col < old_label.at(idx).rings_g.at(row).size() ; ++col ){
                new_label[idx].rings_g[row][col] = old_label.at(idx).rings_g.at(row).at(col);
            }
        }

    }
}




// vtx_set: selected label class
// g: selected graph
bool host_contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}

//      puts into a 2D array the data regarding indexes of rings related to the array of elements
//      2D-array that will contain the result that will be modified
//      1D-array containing idxList
//      1D-array of elements
//      int size of elements
int host_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    int index;
    int idx_list_size= 0 ;


    for( int i = 0; i < elem_size ; ++i){
        index = 0;
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) {idxList[idx_list_size] = index; idx_list_size++;}
            index++;
        }
    }


    for ( int i = 0  ; i < idx_list_size ; ++i  ){
        dim_col[i] = lc->col_ring_size[idxList[i]];
        for ( int j = 0 ; j < dim_col[i] ; ++j ){
            result[i][j] = lc->rings_g[idxList[i]][j];
        }
    }



    return idx_list_size;
}


// return the best select label given an array of labels
void host_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max ;
    for( int i = 0 ; i < lcs_size ; ++i ){
        //printf("LABEL CLASSES INTERNE[%d]\n", i);
        //printLabelClass(lcs[i]);
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            //printf("\nMAX : %d\n", max);
            if( max < min ){
                
                min = max;
                cpyGpuLabelClass(label, lcs[i] );
            }
        }
    }
    return;
}

// compute the bound given a 1D array of struct GpuLabelClass and its size
int host_calc_bound(GpuLabelClass *lcs, int lc_size) {
    int bound = 0;
    for( int i = 0 ; i < lc_size ; ++i){
        if ( lcs[i].g_size > lcs[i].h_size ) bound = bound + lcs[i].h_size;
        else bound = bound + lcs[i].g_size;
    }
    return bound;
}


//return = size of the friends
//friend is the OUTPUT
int host_hoodG(int *friends,int vtx, float edge, float **g, int size_g) {
    int size = 0;
    
    for (int i = 0; i < size_g; i++) {
        if ( g[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }

    return size;
}


// result == size of generated label
// output is l_draft
// input : v
void host_resize(int *array, int size_arr, int place_availabel){
    int count = 0;
    /*bool flag = true;
    for(int i = 0; i < size_arr && flag ; ++i){
        if(array[i] == -1){continue;}
        if( i == count){ count ++; continue; }
        array[count] = array[i]; count++;
        if(count == place_availabel){flag = false;}
    }*/


    for( int i = 0 ; i < size_arr && place_availabel > 0 ; i++){
        if( array[i] != -1 ){
            place_availabel--;
            array[count] = array[i];
            count ++;
        }
    }
}


int host_gen_new_labels(GpuLabelClass *l_draft ,  int v, int w, GpuLabelClass *lcs, int lcs_size, int *idxList) {
    int vs,ws, draft_size = 0;
    int dim_row;
    int count = 0;
    for ( int i = 0 ; i < lcs_size ; ++i ){
       
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize;
            friendsize = host_hoodG( l_draft[draft_size].g , v , gpu_edge_labels[j], gpu_g0 , size_gpu_g0_row);
        
            vs = 0;
            for ( int k = 0; k < friendsize ; ++k ){
                if( host_contains(l_draft[draft_size].g[k] , lcs[i].g, lcs[i].g_size) ){ vs++;  }
                else{ l_draft[draft_size].g[k] = -1;}
            }
        
            host_resize(l_draft[draft_size].g, friendsize, vs );

            dim_row = host_get_ring_match_data(l_draft[draft_size].col_ring_size, l_draft[draft_size].rings_g, idxList ,l_draft[draft_size].g, vs, &lcs[i] );
            

            friendsize = host_hoodG(l_draft[draft_size].h, w, gpu_edge_labels[j], gpu_g1, size_gpu_g1_row );
            //printf("\n esco da hood 2");
            ws = 0;
            for ( int k = 0 ; k < friendsize ; ++k ){
                
                if( host_contains(l_draft[draft_size].h[k], lcs[i].h, lcs[i].h_size) ){  ws++; }
                else {
                    l_draft[draft_size].h[k] = -1;
                }
            }
            host_resize(l_draft[draft_size].h, friendsize, ws );
    
            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) {adj = 1;}
                else { adj = 0; }

                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                draft_size++;
            }
        }
    }
    return draft_size;
}


//given two atoms from the same label, return true if they are matchable, false otherwise
// based on how their rings matches
bool host_matchable(int **v_ring_atoms, int v, int w, GpuLabelClass *lc, int *idxList) {

    host_get_ring_match_data(lc->col_ring_size, v_ring_atoms, idxList , &v, 1 ,lc);
    if( lc->col_ring_size[idxList[0]] > 0  ){
        for(int i = 0; i < lc->col_ring_size[idxList[0]]  ; i++){
            if( v_ring_atoms[0][i] == -1 )return false;
            if( v_ring_atoms[0][i] == w ) return true;
        }
        return false;
    }
    return true;
}


// vtx_set: selected label class
// g: selected graph
void host_select_vertex(int *result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            *result = vtx_set[i];
            *result_pos = i;
        }
    }
    return ;
}



void host_parallel_solve_mcs( ThreadVar *thread_pool_read, ThreadVar *thread_pool_write, int n_threads ){
    
}

void checkError(int iterazione, int line, hipError_t r) {
    if (r != hipSuccess) {
        printf("CUDA error on line %d - iterazione %d : %s\n", line, iterazione,  hipGetErrorString(r));
        exit(0);
    }
}

//copy var1 in var2
void copy_single_ThreadVar(ThreadVar *var2, ThreadVar var1){

    var2->labels_size = var1.labels_size;
    var2->m_size = var1.m_size;

    for (int i = 0; i < var1.labels_size ; i++){
         var2->labels[i].adj =  var1.labels[i].adj;
        var2->labels[i].g_size = var1.labels[i].g_size ;
        var2->labels[i].h_size = var1.labels[i].h_size ;
        var2->labels[i].row_ring_size = var1.labels[i].row_ring_size ; 
        copyIntArray(var2->labels[i].g, var1.labels[i].g,var1.labels[i].g_size );
        copyIntArray(var2->labels[i].h, var1.labels[i].h,var1.labels[i].h_size );
        copyIntArray(var2->labels[i].col_ring_size, var1.labels[i].col_ring_size,var1.labels[i].row_ring_size );

        for(int c = 0; c < 4 ; c++) {
            var2->labels[i].label[c] = var1.labels[i].label[c] ;
        }

        copyIntMatrix(var2->labels[i].rings_g,
                      var1.labels[i].rings_g, 
                      var1.labels[i].row_ring_size,
                      var1.labels[i].col_ring_size ) ;
    }

    for(int i = 0; i < var1.m_size; i++) {
        var2->m_local[i].first =  var1.m_local[i].first;
        var2->m_local[i].second =  var1.m_local[i].second;
    }
}




int autonomouslySolve(ThreadVar *thread_pool, int queue_size, int m_best_size, Pair *auto_pool_m_best, ThreadVar *tmp ){
    

    int flag = 0;
    int z ;
    int iterazione;
    int index;


    while(queue_size > 0) {
        queue_size -- ;
        
        copy_single_ThreadVar(tmp , thread_pool[queue_size] );
        ThreadVar TMP = *tmp;
        GpuLabelClass *label = &TMP.single_label;
        host_select_label(label, TMP.labels, TMP.m_size, TMP.labels_size);



        index = 0;
        flag = 0;
        if((TMP.m_size + host_calc_bound(TMP.labels, TMP.labels_size) < m_best_size) || !label) {flag = 1;}

        if( flag == 0 ){
            
            index = queue_size;
            for( int v_idx = 0 ; v_idx < label->g_size ; ++v_idx){
                for( int w_idx = 0 ; w_idx < label->h_size ; ++w_idx){
                    if( !host_matchable(label->rings_g, label->g[v_idx], label->h[w_idx], label, TMP.idxList ) ) continue;
                    

                    for(z = 0; z < TMP.m_size; z ++){
                        thread_pool[index].m_local[z].first = TMP.m_local[z].first;
                        thread_pool[index].m_local[z].second = TMP.m_local[z].second;
                    } 
                    thread_pool[index].m_size = TMP.m_size +1;
                    thread_pool[index].m_local[z].first = label->g[v_idx];
                    thread_pool[index].m_local[z].second = label->h[v_idx];
                    
                    int l_s = host_gen_new_labels( 
                        thread_pool[index].labels, 
                        label->g[v_idx], 
                        label->h[w_idx] , 
                        TMP.labels,  
                        TMP.labels_size, 
                        TMP.idxList );


                    thread_pool[index].labels_size = l_s;

                    if(thread_pool[index].m_size > m_best_size ){
                        m_best_size = thread_pool[index].m_size;
                        for( int z = 0 ; z < thread_pool[index].m_size; z++ ){
                            auto_pool_m_best[z].first = thread_pool[index].m_local[z].first;
                            auto_pool_m_best[z].second = thread_pool[index].m_local[z].second;
                        }
                    }
                    
                    index ++;
                }
            }
        }


        queue_size = index;
        iterazione ++;
    }

    return m_best_size;
}




void kernel(
                                const std::vector<std::string>& l0,
                                const std::vector<std::string>& l1,
                                vector<queue_elem> Q_filter,
                                int size_initial_label_classes
                                 ) {

    int min_mol_size = std::min(l0.size(), l1.size());
                                    
    //cuda Mallocs
    hipMallocManaged(&m_best_solution , sizeof(Pair)* l1.size());
    //cuda malloc edge labels
    checkError(0, __LINE__ , hipMallocManaged(&gpu_edge_labels, sizeof(float) * edge_labels.size()));
    //cuda malloc adj matrix mol 0
    checkError(0, __LINE__ , hipMallocManaged((void **) &gpu_g0, l0.size() * sizeof(float *)));
    for (int i = 0; i < l0.size(); ++i) { checkError(0, __LINE__ , hipMallocManaged((void **) &(gpu_g0[i]), l0.size() * sizeof(float))); }
    //cuda malloc adj matrix mol 1
    checkError(0, __LINE__ , hipMallocManaged((void **) &gpu_g1, l1.size() * sizeof(float *)));
    for (int i = 0; i < l1.size(); ++i) { checkError(0, __LINE__ , hipMallocManaged((void **) &(gpu_g1[i]), l1.size() * sizeof(float))); }


    //lista di code per ogni singolo thread
    checkError(0, __LINE__ , hipMallocManaged((void **) &thread_pool_list, (Q_filter.size())  * sizeof(ThreadVar *) ));
    checkError(0, __LINE__ , hipMallocManaged(&auto_pool_size, (Q_filter.size()) * sizeof(int *)));
    checkError(0, __LINE__ , hipMallocManaged(&auto_pool_len_m_best, (Q_filter.size()) * sizeof(int *)));
    checkError(0, __LINE__ , hipMallocManaged((void **) &auto_pool_m_best, (Q_filter.size())  * sizeof(Pair *) ));
    checkError(0, __LINE__ , hipMallocManaged( &auto_pool_tmp, (Q_filter.size())  * sizeof(ThreadVar ) ));

    
    for(int i = 0; i < (Q_filter.size()); i++) {
        // Esempi di allocazioni, assicurati di gestire gli errori per ciascuna
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.col_ring_size), sizeof(int) * min_mol_size));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.g), sizeof(int) * l0.size()));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.h), sizeof(int) * l1.size()));
            
            // Allocazione per array di puntatori e iterazione su di essi
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.rings_g), sizeof(int *) * l0.size()));
            for (int h = 0; h < l0.size(); ++h) {
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.rings_g[h]), sizeof(int) * l0.size()));
            }

            // Allocazioni successive
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_m_best[i]), sizeof(Pair) * min_mol_size));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].m_local), sizeof(Pair) * min_mol_size));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].idxList), sizeof(int) * min_mol_size));


            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels), size_initial_label_classes * sizeof(GpuLabelClass)));
            for (int k = 0; k < size_initial_label_classes; ++k) {
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].col_ring_size), sizeof(int) * min_mol_size));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].g), sizeof(int) * l0.size()));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].h), sizeof(int) * l1.size()));

                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].rings_g), sizeof(int *) * l0.size()));
                for (int h = 0; h < l0.size(); ++h) {
                    checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].rings_g[h]), sizeof(int) * l0.size()));
                }
            }

    }


    bool flag = false;
    
    for (int f = 0; f <  Q_filter.size() ; f++) {
        int depth = Q_filter[f].m_local.size();
        int length = (min_mol_size - depth)  ;
        length_list.push_back(length);
        checkError(f, __LINE__, hipMallocManaged((void **)&thread_pool_list[f], sizeof(ThreadVar) * length));

        // Allocazione per le strutture dati all'interno di ciascun ThreadVar
        for (int j = 0; j < length; ++j) {
            // Esempi di allocazioni, assicurati di gestire gli errori per ciascuna
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.col_ring_size), sizeof(int) * min_mol_size));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.g), sizeof(int) * l0.size()));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.h), sizeof(int) * l1.size()));
            
            // Allocazione per array di puntatori e iterazione su di essi
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.rings_g), sizeof(int *) * l0.size()));
            for (int h = 0; h < l0.size(); ++h) {
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.rings_g[h]), sizeof(int) * l0.size()));
            }

            // Allocazioni successive
            checkError(f, __LINE__, hipMallocManaged((void **)&(auto_pool_m_best[f]), sizeof(Pair) * min_mol_size));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].m_local), sizeof(Pair) * min_mol_size));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].idxList), sizeof(int) * min_mol_size));


            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels), size_initial_label_classes * sizeof(GpuLabelClass)));
            for (int k = 0; k < size_initial_label_classes; ++k) {
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].col_ring_size), sizeof(int) * min_mol_size));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].g), sizeof(int) * l0.size()));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].h), sizeof(int) * l1.size()));

                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].rings_g), sizeof(int *) * l0.size()));
                for (int h = 0; h < l0.size(); ++h) {
                    checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].rings_g[h]), sizeof(int) * l0.size()));
                }
            }
        }
    }
//initialize
    //init edge labels
    vectorToPointerEdge(gpu_edge_labels);
    size_edge_labels = edge_labels.size();
    //init adj matrix mol0
    vectorToPointerMatrix(g0, gpu_g0);
    size_gpu_g0_row = g0.size();
    size_gpu_g0_col = g0[0].size();
    //init adj matrix mol 1
    vectorToPointerMatrix(g1, gpu_g1);
    size_gpu_g1_row = g1.size();
    size_gpu_g1_col = g1[0].size();


    //copy the element of the Q_filter inside each QUEUE
    for(int i = 0; i < Q_filter.size(); i++) {
        auto_pool_size[i] = 1;
        auto_pool_len_m_best[i] = m_best.size();
        
        LabelFromCpuToGpu(thread_pool_list[i][0].labels, Q_filter[i].labels );
        thread_pool_list[i][0].labels_size = Q_filter[i].labels.size();
        thread_pool_list[i][0].m_size = Q_filter[i].m_local.size();
        for(int h = 0; h < Q_filter[i].m_local.size(); h++){
            thread_pool_list[i][0].m_local[h].first = Q_filter[i].m_local[h].first;
            thread_pool_list[i][0].m_local[h].second = Q_filter[i].m_local[h].second;
        }
    }

    for(int i = 0; i < Q_filter.size(); i++) {
       auto_pool_len_m_best[i] =  autonomouslySolve(thread_pool_list[i], auto_pool_size[i],auto_pool_len_m_best[i], auto_pool_m_best[i], auto_pool_tmp + i );
    }

    
    for(int i = 0; i < Q_filter.size(); i++) {
       if(auto_pool_len_m_best[i] > m_best.size()){
        m_best.clear();
        pair<int,int> tmp;
        for(int j = 0; j < auto_pool_len_m_best[i] ; j++){
            tmp.first = auto_pool_m_best[i][j].first;
            tmp.second = auto_pool_m_best[i][j].second;
            m_best.push_back(tmp);
        }
       }
    }

    return ;

}

