#include "hip/hip_runtime.h"
//
// Created by davide on 5/3/24.
//

#include "gpu_header.hpp"

#include <vector>
#include <string.h>
#include <string>
#include <queue>
#include <algorithm>
#include <unordered_set>
#include <chrono>  // Include la libreria chrono
using namespace std;

double malloc_elapsed_seconds;



Pair *m_best_solution;



//auto is for autonomous
ThreadVar **thread_pool_list;

int *auto_pool_size;
Pair **auto_pool_m_best;
int *auto_pool_len_m_best;
ThreadVar *auto_pool_tmp;

vector<int> length_list;



__device__ __managed__ float *gpu_edge_labels;
__device__ __managed__ int size_edge_labels;


__device__ __managed__ float **gpu_g0;
__device__ __managed__ int size_gpu_g0_row;
__device__ __managed__ int size_gpu_g0_col;

__device__ __managed__ float **gpu_g1;
__device__ __managed__ int size_gpu_g1_row;
__device__ __managed__ int size_gpu_g1_col;


bool malloc_done = false;



void printLabelClass(GpuLabelClass lb) {
        cout<< lb.label << " [ ";
        cout<< " G("<< lb.g_size << "): ";
        if(!lb.g_size == 0) {for ( int i = 0 ; i < lb.g_size; i++ ) cout<<"["<<lb.g[i]<<"]";}
        cout<< " H("<< lb.h_size << "): ";
        if(!lb.h_size == 0 ) {for ( int i = 0; i <  lb.h_size; i++ ) cout<<"["<<lb.h[i]<<"]";}
        cout<< " RINGS("<< lb.row_ring_size << "): [";
        for( int i = 0; i< lb.row_ring_size; i++){cout<<"("<<lb.col_ring_size[i]<<")"<<"["; for( int j = 0; j <  lb.col_ring_size[i]; j++) cout<<lb.rings_g[i][j]<<", ";  cout<<" ]";}
        cout<<"]";
        cout<< " edge : " <<lb.adj<<" " ;
        cout<< lb.label << " ] "<<endl;
}   
__device__ void copyIntArray(int *a, int *b, int sizeb){
    for ( int i = 0 ; i < sizeb ; i++){
        a[i] = b[i];
    }
}
__device__ void copyIntMatrix(int **a, int **b, int rowsize, int *colsize )
{

    for( int i = 0 ; i < rowsize ; i++){
        for( int j = 0 ; j < colsize[i] ; j++){
            a[i][j] = b[i][j];
        }
    }

}



__device__ void cpyGpuLabelClass(GpuLabelClass *l1, GpuLabelClass l2){
    l1->adj = l2.adj;
    l1->row_ring_size = l2.row_ring_size;
    l1->g_size = l2.g_size;
    l1->h_size = l2.h_size;
    for(int i = 0; i < 4 ; i++) {
        l1->label[i] = l2.label[i];
    }
    //strcpy(l1->label , l2.label);
    copyIntArray( l1->g , l2.g, l2.g_size);
    copyIntArray( l1->h, l2.h, l2.h_size);
    copyIntArray( l1->col_ring_size, l2.col_ring_size , l2.row_ring_size);
    copyIntMatrix( l1->rings_g, l2.rings_g, l2.row_ring_size, l2.col_ring_size );/**/
}

void vectorToPointerEdge(float *gpu_edge_labels){
    if(edge_labels.size() == 0){
        gpu_edge_labels = nullptr;
        return;
    }
    int size = 0;
    for(float edg : edge_labels){
        gpu_edge_labels[size] = edg;
        size++;
    }

    return;
}
void vectorToPointerMatrix(const std::vector<std::vector<float>>& g,float** gpu_g) {
    // Get dimensions of the vector
    int numRows = g.size();
    if (numRows == 0) {
        // Empty vector, set pointers to nullptr
        gpu_g = nullptr;
        return;
    }

    int numCol = g[0].size();

    for (int i = 0; i < numRows; ++i) {
        for (int j = 0; j < numCol; ++j) {
            gpu_g[i][j] = g[i][j];
        }
    }
}

void LabelFromCpuToGpu(GpuLabelClass *new_label, const vector<LabelClass>& old_label ){

    for (int idx = 0 ; idx < old_label.size() ; ++idx ){
        new_label[idx].g_size = old_label.at(idx).g.size();
        new_label[idx].h_size = old_label.at(idx).h.size();
        new_label[idx].row_ring_size = old_label.at(idx).rings_g.size();
        new_label[idx].adj = old_label.at(idx).adj;
        strcpy(new_label[idx].label, old_label.at(idx).label.c_str() );

        for ( int j = 0 ; j < old_label.at(idx).g.size() ; ++j ){
            new_label[idx].g[j] = old_label.at(idx).g.at(j);
        }
        for ( int j = 0 ; j < old_label.at(idx).h.size() ; ++j ){
            new_label[idx].h[j] = old_label.at(idx).h.at(j);}

        for ( int row = 0 ; row < old_label.at(idx).rings_g.size() ; ++row ){
            new_label[idx].col_ring_size[row] = old_label.at(idx).rings_g.at(row).size();
            for ( int col = 0 ; col < old_label.at(idx).rings_g.at(row).size() ; ++col ){
                new_label[idx].rings_g[row][col] = old_label.at(idx).rings_g.at(row).at(col);
            }
        }

    }
}




// vtx_set: selected label class
// g: selected graph
__device__ bool device_contains(int value, int *arr, int size) {
    for (int i = 0; i < size; ++i) {
        if (arr[i] == value) {
            return true;
        }
    }
    return false;
}

//      puts into a 2D array the data regarding indexes of rings related to the array of elements
//      2D-array that will contain the result that will be modified
//      1D-array containing idxList
//      1D-array of elements
//      int size of elements
__device__ int device_get_ring_match_data(int *dim_col, int **result, int *idxList, int *elems, int elem_size, GpuLabelClass *lc){
    int index;
    int idx_list_size= 0 ;


    for( int i = 0; i < elem_size ; ++i){
        index = 0;
        for( int j = 0 ; j < lc->g_size ; ++j ){
            if( lc->g[j] == elems[i] ) {idxList[idx_list_size] = index; idx_list_size++;}
            index++;
        }
    }


    for ( int i = 0  ; i < idx_list_size ; ++i  ){
        dim_col[i] = lc->col_ring_size[idxList[i]];
        for ( int j = 0 ; j < dim_col[i] ; ++j ){
            result[i][j] = lc->rings_g[idxList[i]][j];
        }
    }



    return idx_list_size;
}


// return the best select label given an array of labels
__device__ void device_select_label(GpuLabelClass *label , GpuLabelClass *lcs, int map_size, int lcs_size){
    int min = 999;
    int max ;
    for( int i = 0 ; i < lcs_size ; ++i ){
        //printf("LABEL CLASSES INTERNE[%d]\n", i);
        //printLabelClass(lcs[i]);
        if( lcs[i].adj == 1 || map_size == 0 ){
            if( lcs[i].g_size > lcs[i].h_size ) max = lcs[i].g_size;
            else max = lcs[i].h_size;
            //printf("\nMAX : %d\n", max);
            if( max < min ){
                
                min = max;
                cpyGpuLabelClass(label, lcs[i] );
            }
        }
    }
    return;
}

// compute the bound given a 1D array of struct GpuLabelClass and its size
__device__ int device_calc_bound(GpuLabelClass *lcs, int lc_size) {
    int bound = 0;
    for( int i = 0 ; i < lc_size ; ++i){
        if ( lcs[i].g_size > lcs[i].h_size ) bound = bound + lcs[i].h_size;
        else bound = bound + lcs[i].g_size;
    }
    return bound;
}


//return = size of the friends
//friend is the OUTPUT
__device__ int device_hoodG(int *friends,int vtx, float edge, float **g, int size_g) {
    int size = 0;
    
    for (int i = 0; i < size_g; i++) {
        if ( g[i][vtx] == edge && vtx != i) {
            friends[size] = i;
            size++;
        }
    }

    return size;
}


// result == size of generated label
// output is l_draft
// input : v
__device__ void device_resize(int *array, int size_arr, int place_availabel){
    int count = 0;

    for( int i = 0 ; i < size_arr && place_availabel > 0 ; i++){
        if( array[i] != -1 ){
            place_availabel--;
            array[count] = array[i];
            count ++;
        }
    }
}


__device__ int device_gen_new_labels(GpuLabelClass *l_draft ,  int v, int w, GpuLabelClass *lcs, int lcs_size, int *idxList) {
    int vs,ws, draft_size = 0;
    int dim_row;
    int count = 0;
    for ( int i = 0 ; i < lcs_size ; ++i ){
      // printf("\ndevice_gen_new_labels iterazione num : %d", i );
        for ( int j = 0 ; j < size_edge_labels ; ++j ){
            int friendsize;
            friendsize = device_hoodG( l_draft[draft_size].g , v , gpu_edge_labels[j], gpu_g0 , size_gpu_g0_row);
        
            vs = 0;
            for ( int k = 0; k < friendsize ; ++k ){
                if( device_contains(l_draft[draft_size].g[k] , lcs[i].g, lcs[i].g_size) ){ vs++;  }
                else{ l_draft[draft_size].g[k] = -1;}
            }
        
            device_resize(l_draft[draft_size].g, friendsize, vs );

            dim_row = device_get_ring_match_data(l_draft[draft_size].col_ring_size, l_draft[draft_size].rings_g, idxList ,l_draft[draft_size].g, vs, &lcs[i] );
            

            friendsize = device_hoodG(l_draft[draft_size].h, w, gpu_edge_labels[j], gpu_g1, size_gpu_g1_row );
            //printf("\n esco da hood 2");
            ws = 0;
            for ( int k = 0 ; k < friendsize ; ++k ){
                
                if( device_contains(l_draft[draft_size].h[k], lcs[i].h, lcs[i].h_size) ){  ws++; }
                else {
                    l_draft[draft_size].h[k] = -1;
                }
            }
            device_resize(l_draft[draft_size].h, friendsize, ws );
    
            int adj;
            if ( ws > 0 && vs > 0 ){
                if( gpu_edge_labels[j] != 0.0 || lcs[i].adj == 1 ) {adj = 1;}
                else { adj = 0; }

                l_draft[draft_size].g_size = vs;
                l_draft[draft_size].h_size = ws;
                l_draft[draft_size].row_ring_size = dim_row;
                l_draft[draft_size].adj = adj;
                for( int c = 0 ; c < 4 ; c++){
                    l_draft[draft_size].label[c] = lcs[i].label[c];
                }
                draft_size++;
            }
        }
    }
    return draft_size;
}


//given two atoms from the same label, return true if they are matchable, false otherwise
// based on how their rings matches
__device__ bool device_matchable(int **v_ring_atoms, int v, int w, GpuLabelClass *lc, int *idxList) {

    device_get_ring_match_data(lc->col_ring_size, v_ring_atoms, idxList , &v, 1 ,lc);
    if( lc->col_ring_size[idxList[0]] > 0  ){
        for(int i = 0; i < lc->col_ring_size[idxList[0]]  ; i++){
            if( v_ring_atoms[0][i] == -1 )return false;
            if( v_ring_atoms[0][i] == w ) return true;
        }
        return false;
    }
    return true;
}


// vtx_set: selected label class
// g: selected graph
__device__ void device_select_vertex(int *result, int *result_pos, int *vtx_set, int vtx_size, float **g, int num_row, int num_column) {
    int max_deg = -1;
    int vtx = 0;

    for(int i = 0; i < vtx_size; i++){
        int deg = 0;
        for(int j = 0; j < num_column; j++){
            int consider = g[vtx_set[i]][j];
            if(consider != 0){
                deg++;
            }
        }

        if(deg>max_deg){
            max_deg = deg;
            *result = vtx_set[i];
            *result_pos = i;
        }
    }
    return ;
}




void checkError(int iterazione, int line, hipError_t r) {
    if (r != hipSuccess) {
        printf("CUDA error on line %d - iterazione %d : %s\n", line, iterazione,  hipGetErrorString(r));
        exit(0);
    }
}

//copy var1 in var2
__device__ void copy_single_ThreadVar(ThreadVar *var2, ThreadVar var1){

    var2->labels_size = var1.labels_size;
    var2->m_size = var1.m_size;

    for (int i = 0; i < var1.labels_size ; i++){
         var2->labels[i].adj =  var1.labels[i].adj;
        var2->labels[i].g_size = var1.labels[i].g_size ;
        var2->labels[i].h_size = var1.labels[i].h_size ;
        var2->labels[i].row_ring_size = var1.labels[i].row_ring_size ; 
        copyIntArray(var2->labels[i].g, var1.labels[i].g,var1.labels[i].g_size );
        copyIntArray(var2->labels[i].h, var1.labels[i].h,var1.labels[i].h_size );
        copyIntArray(var2->labels[i].col_ring_size, var1.labels[i].col_ring_size,var1.labels[i].row_ring_size );

        for(int c = 0; c < 4 ; c++) {
            var2->labels[i].label[c] = var1.labels[i].label[c] ;
        }

        copyIntMatrix(var2->labels[i].rings_g,
                      var1.labels[i].rings_g, 
                      var1.labels[i].row_ring_size,
                      var1.labels[i].col_ring_size ) ;
    }

    for(int i = 0; i < var1.m_size; i++) {
        var2->m_local[i].first =  var1.m_local[i].first;
        var2->m_local[i].second =  var1.m_local[i].second;
    }
}



__device__ void print_labels(GpuLabelClass *labels, int labels_size){
    for(int i = 0; i < labels_size; i++){
        printf("\nLabel : ");
        for(int j = 0; j < 4; j++){
            printf("%c ", labels[i].label[j] );
        }
        printf("\n");


        printf("\ng_size : %d    h_size : %d", labels[i].g_size, labels[i].h_size );

    }
}

__global__ void autonomouslySolve(ThreadVar **thread_pool_list, int *queue_size_list, int* m_best_size,Pair **auto_pool_m_best_list, ThreadVar *tmp, int Q_size ){
    
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalIdx >= Q_size) return;

    ThreadVar *thread_pool = thread_pool_list[globalIdx];
    int queue_size = queue_size_list[globalIdx];
    int m_best_size_index = globalIdx;
    Pair *auto_pool_m_best = auto_pool_m_best_list[globalIdx];
    tmp = tmp + globalIdx;



    int flag = 0;
    int z ;
    int iterazione;
    int index;

    int max_legth_queue = 1;

    //mi prendo il primo valore della coda 
    copy_single_ThreadVar(tmp , thread_pool[0] );

  
    printf("\nautonomouslySolve globalIdx :  %d \nlabels_size : %d", globalIdx, tmp -> labels_size);  
   //print_labels(tmp->labels, tmp->labels_size);

    while(queue_size > 0) {
        
        if(queue_size > max_legth_queue) {
            max_legth_queue = queue_size;
        }


        queue_size -- ;
        
        copy_single_ThreadVar(tmp , thread_pool[queue_size] );
        ThreadVar TMP = *tmp;
        GpuLabelClass *label = &TMP.single_label;
        device_select_label(label, TMP.labels, TMP.m_size, TMP.labels_size);



        index = 0;
        flag = 0;
        if((TMP.m_size + device_calc_bound(TMP.labels, TMP.labels_size) < m_best_size[m_best_size_index]) || !label) {flag = 1;}

        if( flag == 0 ){
            
            index = queue_size;
            for( int v_idx = 0 ; v_idx < label->g_size ; ++v_idx){
                for( int w_idx = 0 ; w_idx < label->h_size ; ++w_idx){
                    if( !device_matchable(label->rings_g, label->g[v_idx], label->h[w_idx], label, TMP.idxList ) ) continue;
                    

                    for(z = 0; z < TMP.m_size; z ++){
                        thread_pool[index].m_local[z].first = TMP.m_local[z].first;
                        thread_pool[index].m_local[z].second = TMP.m_local[z].second;
                    } 
                    thread_pool[index].m_size = TMP.m_size +1;
                    thread_pool[index].m_local[z].first = label->g[v_idx];
                    thread_pool[index].m_local[z].second = label->h[v_idx];
                    
                    int l_s = device_gen_new_labels( 
                        thread_pool[index].labels, 
                        label->g[v_idx], 
                        label->h[w_idx] , 
                        TMP.labels,  
                        TMP.labels_size, 
                        TMP.idxList );


                    thread_pool[index].labels_size = l_s;

                    if(thread_pool[index].m_size > m_best_size[m_best_size_index] ){
                        m_best_size[m_best_size_index] = thread_pool[index].m_size;
                        for( int z = 0 ; z < thread_pool[index].m_size; z++ ){
                            auto_pool_m_best[z].first = thread_pool[index].m_local[z].first;
                            auto_pool_m_best[z].second = thread_pool[index].m_local[z].second;
                        }
                    }
                    
                    index ++;
                }
            }
        }


        queue_size = index;
        iterazione ++;
    }

   return ;
}


void malloc( vector<queue_elem> Q_filter, int min_mol_size){
    clock_t start = clock();                          
    //cuda Mallocs
    hipMallocManaged(&m_best_solution , sizeof(Pair)* l1.size());
    //cuda malloc edge labels
    checkError(0, __LINE__ , hipMallocManaged(&gpu_edge_labels, sizeof(float) * edge_labels.size()));
    //cuda malloc adj matrix mol 0
    checkError(0, __LINE__ , hipMallocManaged((void **) &gpu_g0, l0.size() * sizeof(float *)));
    for (int i = 0; i < l0.size(); ++i) { checkError(0, __LINE__ , hipMallocManaged((void **) &(gpu_g0[i]), l0.size() * sizeof(float))); }
    //cuda malloc adj matrix mol 1
    checkError(0, __LINE__ , hipMallocManaged((void **) &gpu_g1, l1.size() * sizeof(float *)));
    for (int i = 0; i < l1.size(); ++i) { checkError(0, __LINE__ , hipMallocManaged((void **) &(gpu_g1[i]), l1.size() * sizeof(float))); }


    //lista di code per ogni singolo thread
    checkError(0, __LINE__ , hipMallocManaged((void **) &thread_pool_list, (Q_filter.size())  * sizeof(ThreadVar *) ));
    checkError(0, __LINE__ , hipMallocManaged(&auto_pool_size, (Q_filter.size()) * sizeof(int *)));
    checkError(0, __LINE__ , hipMallocManaged(&auto_pool_len_m_best, (Q_filter.size()) * sizeof(int *)));
    checkError(0, __LINE__ , hipMallocManaged((void **) &auto_pool_m_best, (Q_filter.size())  * sizeof(Pair *) ));
    checkError(0, __LINE__ , hipMallocManaged( &auto_pool_tmp, (Q_filter.size())  * sizeof(ThreadVar ) ));

    int common_queue_element_size = 3;
    
    for(int i = 0; i < (Q_filter.size()); i++) {
        // Esempi di allocazioni, assicurati di gestire gli errori per ciascuna
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.col_ring_size), sizeof(int) * min_mol_size));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.g), sizeof(int) * max_first_len));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.h), sizeof(int) * max_first_len));
            
            // Allocazione per array di puntatori e iterazione su di essi
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.rings_g), sizeof(int *) * max_first_len));
            for (int h = 0; h < l0.size(); ++h) {
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].single_label.rings_g[h]), sizeof(int) * max_first_len));
            }

            // Allocazioni successive
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_m_best[i]), sizeof(Pair) * min_mol_size));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].m_local), sizeof(Pair) * min_mol_size));
            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].idxList), sizeof(int) * min_mol_size));


            checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels), ( size_initial_label_classes + (2 * size_edge_labels)) * sizeof(GpuLabelClass)));

            //first + edge_labels
            for(int s = 0; s < size_edge_labels ; s++){
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].col_ring_size), sizeof(int) * max_first_len));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].g), sizeof(int) * max_first_len));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].h), sizeof(int) * max_first_len));

                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].rings_g), sizeof(int *) * max_first_len));
                for (int h = 0; h < max_first_len ; ++h) {
                    checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].rings_g[h]), sizeof(int) * max_first_len));
                }
            }

            //second + edge_labels
            for(int s = size_edge_labels; s < (2 * size_edge_labels); s++ ){
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].col_ring_size), sizeof(int) * max_second_len));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].g), sizeof(int) * max_second_len));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].h), sizeof(int) * max_second_len));

                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].rings_g), sizeof(int *) * max_second_len));
                for (int h = 0; h < max_second_len ; ++h) {
                    checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[s].rings_g[h]), sizeof(int) * max_second_len));
                }
            }


            // 5 is an indicative number

            for (int k = (2 * size_edge_labels); k < size_initial_label_classes + (2 * size_edge_labels); ++k) {
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].col_ring_size), sizeof(int) * common_queue_element_size));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].g), sizeof(int) * common_queue_element_size));
                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].h), sizeof(int) * common_queue_element_size));

                checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].rings_g), sizeof(int *) * common_queue_element_size));
                for (int h = 0; h < 5; ++h) {
                    checkError(i, __LINE__, hipMallocManaged(&(auto_pool_tmp[i].labels[k].rings_g[h]), sizeof(int) * common_queue_element_size));
                }
            }

    }


    bool flag = false;
    
    for (int f = 0; f <  Q_filter.size() ; f++) {
        int depth = Q_filter[f].m_local.size();
        int length = (min_mol_size - depth) / 2  ;
        length_list.push_back(length);


        checkError(f, __LINE__, hipMallocManaged((void **)&thread_pool_list[f], sizeof(ThreadVar) * length));

        // Allocazione per le strutture dati all'interno di ciascun ThreadVar
        // prova
        //length = 15;

        for (int j = 0; j < length; ++j) {
            // Esempi di allocazioni, assicurati di gestire gli errori per ciascuna
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.col_ring_size), sizeof(int) * max_first_len));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.g), sizeof(int) * max_first_len));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.h), sizeof(int) * max_first_len));
            
            // Allocazione per array di puntatori e iterazione su di essi
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.rings_g), sizeof(int *) * max_first_len));
            for (int h = 0; h < max_first_len; ++h) {
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].single_label.rings_g[h]), sizeof(int) * max_first_len));
            }

            // Allocazioni successive
            checkError(f, __LINE__, hipMallocManaged((void **)&(auto_pool_m_best[f]), sizeof(Pair) * min_mol_size));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].m_local), sizeof(Pair) * min_mol_size));
            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].idxList), sizeof(int) * min_mol_size));


            checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels), (size_initial_label_classes  + (2 * size_edge_labels) ) * sizeof(GpuLabelClass)));
            
            //first + edge_label
            for(int s = 0; s < size_edge_labels ; s++){
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].col_ring_size), sizeof(int) * max_first_len));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].g), sizeof(int) * max_first_len));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].h), sizeof(int) * max_first_len));

                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].rings_g), sizeof(int *) * max_first_len));
                for (int h = 0; h < max_first_len; ++h) {
                    checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].rings_g[h]), sizeof(int) * max_first_len));
                }
            }

            //second + edge_labels
            for( int s = size_edge_labels ; s < (2 * size_edge_labels); s++){
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].col_ring_size), sizeof(int) * max_second_len));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].g), sizeof(int) * max_second_len));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].h), sizeof(int) * max_second_len));

                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].rings_g), sizeof(int *) * max_second_len));
                for (int h = 0; h < max_second_len; ++h) {
                    checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[s].rings_g[h]), sizeof(int) * max_second_len));
                }
            }

            for (int k = ((2 * size_edge_labels)); k < size_initial_label_classes + (2 * size_edge_labels) ; ++k) {
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].col_ring_size), sizeof(int) * common_queue_element_size));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].g), sizeof(int) * common_queue_element_size));
                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].h), sizeof(int) * common_queue_element_size));

                checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].rings_g), sizeof(int *) * common_queue_element_size));
                for (int h = 0; h < 5; ++h) {
                    checkError(f, __LINE__, hipMallocManaged((void **)&(thread_pool_list[f][j].labels[k].rings_g[h]), sizeof(int) * common_queue_element_size));
                }
            }
        }
    }
    clock_t end = clock();
     // Calculate elapsed time in seconds
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;
    malloc_elapsed_seconds = elapsed_seconds;
      // Print the elapsed time in seconds
    std::cout << "\nMALLOC Elapsed time: " << elapsed_seconds << " seconds" << std::endl;

}



void kernel( vector<queue_elem> Q_filter  ) {

    int min_mol_size = std::min(l0.size(), l1.size());

    printf("\nmin_mol_size : %d", min_mol_size);
    printf("\nsize_initial_label_classes : %d", size_initial_label_classes);
    cout << "\nfirst max size : " << max_first_len << " second max size : " << max_second_len << endl;
    size_edge_labels = edge_labels.size();
    cout << "\nsize_edge_labels : " << size_edge_labels << endl ;

    if(! malloc_done){
        malloc(Q_filter, min_mol_size);
        malloc_done = true;
    }
    
    //initialize
    //init edge labels
    vectorToPointerEdge(gpu_edge_labels);
   
    //init adj matrix mol0
    vectorToPointerMatrix(g0, gpu_g0);
    size_gpu_g0_row = g0.size();
    size_gpu_g0_col = g0[0].size();
    //init adj matrix mol 1
    vectorToPointerMatrix(g1, gpu_g1);
    size_gpu_g1_row = g1.size();
    size_gpu_g1_col = g1[0].size();





    //copy the element of the Q_filter inside each QUEUE
    for(int i = 0; i < Q_filter.size(); i++) {
        auto_pool_size[i] = 1;
        auto_pool_len_m_best[i] = m_best.size();
        
        LabelFromCpuToGpu(thread_pool_list[i][0].labels, Q_filter[i].labels );
        thread_pool_list[i][0].labels_size = Q_filter[i].labels.size();
        thread_pool_list[i][0].m_size = Q_filter[i].m_local.size();
        for(int h = 0; h < Q_filter[i].m_local.size(); h++){
            thread_pool_list[i][0].m_local[h].first = Q_filter[i].m_local[h].first;
            thread_pool_list[i][0].m_local[h].second = Q_filter[i].m_local[h].second;
        }
    }

    
    autonomouslySolve<<<32,32>>>(thread_pool_list, auto_pool_size ,auto_pool_len_m_best, auto_pool_m_best, auto_pool_tmp ,Q_filter.size());
     // Attendi il completamento del kernel
    hipDeviceSynchronize();

    
    for(int i = 0; i < Q_filter.size(); i++) {
       if(auto_pool_len_m_best[i] > m_best.size()){
        m_best.clear();
        pair<int,int> tmp;
        for(int j = 0; j < auto_pool_len_m_best[i] ; j++){
            tmp.first = auto_pool_m_best[i][j].first;
            tmp.second = auto_pool_m_best[i][j].second;
            m_best.push_back(tmp);
        }
       }
    }

    return ;

}

