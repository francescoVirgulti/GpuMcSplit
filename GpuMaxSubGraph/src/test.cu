#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//

#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include <ctime>
#include <chrono>  // Include la libreria chrono

using namespace std;
using namespace RDKit;



int main()
{

    int *pointer_tmp;
    hipMallocManaged(&pointer_tmp, sizeof(int) * 2);

    
    string s0 = "Fc1cc(NCCN2C=Cn3nc(cc3C2=O)c4occc4)ccc1C(=O)CC5CCC5";
    string s1 = "O=C(CC1CCC1)c2ccc(NCCN3C=Cn4nc(cc4C3=O)c5occc5)cc2";


    clock_t start = clock();
    ROMol result = smiles_mcs(s0, s1 );
    clock_t end = clock();

    
    // Calculate elapsed time in seconds
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    std::vector<std::string> result_string;
    for (const auto &atom : result.atoms()) {
        result_string.push_back(atom->getSymbol());
    }


    cout << "[";
    for ( int idx = 0; idx < result_string.size(); idx++ ){
        if(idx == result_string.size()-1 ){
            cout <<"'"<<result_string.at(idx)<<"']"<<endl;
        }
        else cout <<"'"<<result_string.at(idx)<<"', ";
    }
    cout<<"done\n\n";

     // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;
    std::cout << "\nElapsed time [WITHOUT MALLOC]: " << elapsed_seconds - malloc_elapsed_seconds << " seconds" << std::endl;

    return 0;
}
