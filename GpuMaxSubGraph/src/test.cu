#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//

#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include <ctime>
#include <chrono>  // Include la libreria chrono

using namespace std;
using namespace RDKit;

int max_l0_size = 0;
int max_l1_size = 0; 
int max_first_len_initialized = 0;

int edge_label_size = 4;
int max_initial_label_size = 0;
bool state_initialized = true;




int main()
{

    int *pointer_tmp;
    hipMallocManaged(&pointer_tmp, sizeof(int) * 2);

    
  string s0 = "CC(C)[C@@H]1CC[C@@H](C)C[C@H]1OC(=O)c2ccccc2c3c(C)cccc3CN(C)C4CCCCC4";
    string s1 = "O=C(OC1CCCCC1)c2ccccc2c3ccccc3CNC4CCCCC4";
 ROMol result = smiles_mcs(s0, s1 );
 state_initialized = false;

    clock_t start = clock();
     result = smiles_mcs(s0, s1 );
    clock_t end = clock();

    
    // Calculate elapsed time in seconds
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    std::vector<std::string> result_string;
    for (const auto &atom : result.atoms()) {
        result_string.push_back(atom->getSymbol());
    }


    cout << "[";
    for ( int idx = 0; idx < result_string.size(); idx++ ){
        if(idx == result_string.size()-1 ){
            cout <<"'"<<result_string.at(idx)<<"']"<<endl;
        }
        else cout <<"'"<<result_string.at(idx)<<"', ";
    }
    cout<<"done\n\n";

     // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;
    std::cout << "\nElapsed time [WITHOUT MALLOC]: " << elapsed_seconds - malloc_elapsed_seconds << " seconds" << std::endl;

    return 0;
}
