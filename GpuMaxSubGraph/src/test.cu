#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//

#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include <ctime>
#include <chrono>  // Include la libreria chrono

using namespace std;
using namespace RDKit;



int main()
{

    /*std::string filename = "smiles.txt";
    std::vector<std::string> smiles;
    std::ifstream file(filename);

    if (file.is_open()) {
    std::string line;
        while (std::getline(file, line)) {
            smiles.push_back(line);
        }

    } else {
    std::cerr << "Error opening file: " << filename << std::endl;
  }

  

    // Close the file
    file.close();

  std::ofstream outfile("output.txt");
  std::streambuf* original_cout_buffer = std::cout.rdbuf();  // Save original buffer
  std::cout.rdbuf(outfile.rdbuf());
    

    

    clock_t start = clock();
    ROMol result;
    //cout<<"PRE FUNCTION" ;
    for ( int i = 0 ; i<smiles.size() -1; ++i) {
        for(int j = i+1; j < smiles.size(); j++){
            result = smiles_mcs(smiles.at(i), smiles.at(j), 1,1);

            std::vector<std::string> result_string;
            for (const auto &atom : result.atoms()) {
                result_string.push_back(atom->getSymbol());
            }
            cout << "[";
            for ( int idx = 0; idx < result_string.size(); idx++ ){
                if(idx == result_string.size()-1 ){
                    cout <<"'"<<result_string.at(idx)<<"']"<<endl;
                }
                else cout <<"'"<<result_string.at(idx)<<"', ";
            }
        } 
    }

    clock_t end = clock();

    // Calculate elapsed time in seconds
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;


std::cout.rdbuf(original_cout_buffer);*/
    int *pointer_tmp;
    hipMallocManaged(&pointer_tmp, sizeof(int) * 2);

    string s0 = "COCCCOc1cc(C[C@@H](C[C@H](NC(=O)OC(C)(C)C)C(O)CCCC(=O)N2CC3CCC(C3)C2)C(C)C)ccc1OC";
    string s1 = "O=C(CCCCCCCCc1ccccc1)N2CC3CCC(C3)C2";

    clock_t start = clock();
    ROMol result = smiles_mcs(s0, s1 );
    clock_t end = clock();

    
    // Calculate elapsed time in seconds
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    std::vector<std::string> result_string;
    for (const auto &atom : result.atoms()) {
        result_string.push_back(atom->getSymbol());
    }


    cout << "[";
    for ( int idx = 0; idx < result_string.size(); idx++ ){
        if(idx == result_string.size()-1 ){
            cout <<"'"<<result_string.at(idx)<<"']"<<endl;
        }
        else cout <<"'"<<result_string.at(idx)<<"', ";
    }
    cout<<"done\n\n";

     // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;

    return 0;
}
