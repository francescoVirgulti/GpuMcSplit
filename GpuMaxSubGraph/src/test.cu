#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//
#include <rdkit/GraphMol/GraphMol.h>
#include <GraphMol/SmilesParse/SmilesParse.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include "cuda_header.h"
#include <ctime>
#include <chrono>  // Include la libreria chrono

using namespace std;
using namespace RDKit;

int max_l0_size = 0;
int max_l1_size = 0; 
int max_first_len_initialized = 0;

int edge_label_size = 4;
int max_initial_label_size = 0;
bool state_initialized = true;


//auto is for autonomous
ThreadVar **thread_pool_list; 
int *auto_pool_size;
Pair **auto_pool_m_best;
int *auto_pool_len_m_best;
ThreadVar *auto_pool_tmp;

vector<int> length_list;

Pair *m_best_solution;

float *main_gpu_edge_labels;
float **main_gpu_g0;
float **main_gpu_g1;

bool malloc_done = false;


int main()
{

    int *pointer_tmp;
    hipMallocManaged(&pointer_tmp, sizeof(int) * 2);

      std::vector<std::pair<std::string, std::string>> molecules;

        std::string filename = "input.txt";
        std::string outputFilename = "output.txt";
        // Open the output file for writing
        std::ofstream outputFile(outputFilename);
        if (!outputFile) {
            std::cerr << "Error opening output file: " << outputFilename << std::endl;
            return 1;
        }
        //std::vector<std::pair<std::string, std::string>> smiles;
        std::ifstream file(filename);
        int skip;
        std::string first, second;
        ROMol result;
        int i = 0;


        if (file.is_open()) {
            std::string line;
            while (std::getline(file, line) ) {

                pair<string,string> tmp;
                std::istringstream iss(line);
                
                // Skip the first int
                iss >> skip;
                // Read the first string
                iss >> first;
                // Skip the next four ints
                for (int i = 0; i < 4; ++i) {
                    iss >> skip;
                }
                // Read the second string
                iss >> second; 
                clock_t start = clock();
                // Store the pair of strings
                tmp.first = first;
                tmp.second = second;
                molecules.push_back(tmp);
                
                i++;
            }
        } else {
            std::cerr << "Error opening file: " << filename << std::endl;
        }

        cout << "molecules size : " << molecules.size() <<endl;


        clock_t start = clock();
        for(pair mol_pair : molecules ){
            smiles_mcs(mol_pair.first, mol_pair.second);
        }
        clock_t end = clock();
        state_initialized = false;
         double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;


        std::vector<std::string> result_string;
        int index = 0;
        for(pair mol_pair : molecules ){
            cout << "__________________________MOLECULES PAIR NUM : " << index << endl;
            index++;
            result.clear();
            RWMol mol0 = *SmilesToMol(mol_pair.first);
            RWMol mol1 = *SmilesToMol(mol_pair.second); 
            clock_t start = clock();
            result = mol_mcs(mol0, mol1, 1,1,0);
            clock_t end = clock();
            elapsed_seconds = elapsed_seconds + (double)(end - start) / CLOCKS_PER_SEC;
            result_string.clear();
            for (const auto &atom : result.atoms()) {
                result_string.push_back(atom->getSymbol());
            }

                outputFile << "[";
                for ( int idx = 0; idx < result_string.size(); idx++ ){
                    if(idx == result_string.size()-1 ){
                        outputFile <<"'"<<result_string.at(idx)<<"']" << endl;;
                    }
                    else outputFile <<"'"<<result_string.at(idx)<<"', ";
                }
        }

    




    
    
    // Calculate elapsed time in seconds
    //double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    



     // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;
    std::cout << "\n MALLOC Elapsed time: " << malloc_elapsed_seconds << " seconds" << std::endl;
    std::cout << "\nElapsed time [WITHOUT MALLOC]: " << elapsed_seconds - malloc_elapsed_seconds << " seconds" << std::endl;

    return 0;
}
