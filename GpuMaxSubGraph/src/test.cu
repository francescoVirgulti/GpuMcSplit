#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//
#include <rdkit/GraphMol/GraphMol.h>
#include <GraphMol/SmilesParse/SmilesParse.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include "cuda_header.h"
#include <ctime>
#include <chrono>  

using namespace std;
using namespace RDKit;

int algorithm_used;

// GLOBAL VARIABLE
int max_l0_size = 0;
int max_l1_size = 0; 
int max_first_len_initialized = 0;
int edge_label_size = 4;
int max_initial_label_size = 0;
bool state_initialized = true;
// --- auto : autonomous 
ThreadVar **thread_pool_list; 
int *auto_pool_size;
Pair **auto_pool_m_best;
int *auto_pool_len_m_best;
ThreadVar *auto_pool_tmp;

vector<int> length_list;
Pair *m_best_solution;
float *main_gpu_edge_labels;
float **main_gpu_g0;
float **main_gpu_g1;

bool malloc_done = false;


int main() {

    int *pointer_tmp;
    hipMallocManaged(&pointer_tmp, sizeof(int) * 2);
    std::vector<std::pair<std::string, std::string>> molecules;
    std::string filename = "input.txt";
    std::string outputFilename = "output.txt";



    // Display menu
    std::cout << "Which algorithm would you like to use?" << std::endl;
    std::cout << "1 - mcSplit" << std::endl;
    std::cout << "2 - iter_mcSplit" << std::endl;
    std::cout << "3 - gpu_mcSplit" << std::endl;
    
    // Get user input
    std::cout << "Enter your choice (1, 2, or 3): ";
    std::cin >> algorithm_used;


    // Open the output file for writing
    std::ofstream outputFile(outputFilename);
    if (!outputFile) {
        std::cerr << "Error opening output file: " << outputFilename << std::endl;
        return 1;
    }

    std::ifstream file(filename);
    int skip;
    std::string first, second;
    ROMol result;
    int i = 0;


    if (file.is_open()) {
        std::string line;
        while (std::getline(file, line) ) {

            pair<string,string> tmp;
            std::istringstream iss(line);
            
            // Skip the first int
            iss >> skip;
            // Read the first string
            iss >> first;
            // Skip the next four ints
            for (int i = 0; i < 4; ++i) {
                iss >> skip;
            }
            // Read the second string
            iss >> second; 
            clock_t start = clock();
            // Store the pair of strings
            tmp.first = first;
            tmp.second = second;
            molecules.push_back(tmp);
            
            i++;
        }
    } else {
        std::cerr << "Error opening file: " << filename << std::endl;
    }

    cout << "molecules size : " << molecules.size() <<endl;

    // Define the exact amount of space needed
    clock_t start = clock();
    if(algorithm_used == 3 ){
        for(pair mol_pair : molecules ){
            smiles_mcs(mol_pair.first, mol_pair.second);
        }
        state_initialized = false;
    }
    clock_t end = clock();
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;


    std::vector<std::string> result_string;
    int index = 0;

    for(pair mol_pair : molecules ){
        cout << "__________________________MOLECULES PAIR NUM : " << index << endl;
        index++;
        result.clear();
        RWMol mol0 = *SmilesToMol(mol_pair.first);
        RWMol mol1 = *SmilesToMol(mol_pair.second); 
        clock_t start = clock();
        result = mol_mcs(mol0, mol1, 1,1,0);
        clock_t end = clock();
        elapsed_seconds = elapsed_seconds + (double)(end - start) / CLOCKS_PER_SEC;
        result_string.clear();
        for (const auto &atom : result.atoms()) {
            result_string.push_back(atom->getSymbol());
        }
        std::sort(result_string.begin(), result_string.end());
        if(index < 2872){
            outputFile << "[";
            for ( int idx = 0; idx < result_string.size(); idx++ ){
                if(idx == result_string.size()-1 ){
                    outputFile <<"'"<<result_string.at(idx)<<"']" << endl;;
                }
                else outputFile <<"'"<<result_string.at(idx)<<"', ";
            }
        }
    }








    // Calculate elapsed time in seconds
    //double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;
    std::cout << "\nMALLOC Elapsed time: " << malloc_elapsed_seconds << " seconds" << std::endl;
    std::cout << "\nElapsed time [WITHOUT MALLOC]: " << elapsed_seconds - malloc_elapsed_seconds << " seconds" << std::endl;

    return 0;
}
