#include "hip/hip_runtime.h"
//
// Created by davide on 4/19/24.
//

#include <fstream>
#include <iostream>
#include <vector>
#include <string>
#include "test.hpp"
#include <ctime>
#include <chrono>  // Include la libreria chrono

using namespace std;
using namespace RDKit;



int main()
{

    int *pointer_tmp;
    hipMallocManaged(&pointer_tmp, sizeof(int) * 2);

    
    string s0 = "[O-][N+](=O)c1cccc(Nc2nc(c3ccccc3)c4cc(NCCN5C=Cn6nc(cc6C5=O)c7occc7)ccc4n2)c1";
    string s1 = "O=C1N(CCNc2ccc3nc(Nc4ccccc4)nc(c5ccccc5)c3c2)C=Cn6nc(cc16)c7occc7";


    clock_t start = clock();
    ROMol result = smiles_mcs(s0, s1 );
    clock_t end = clock();

    
    // Calculate elapsed time in seconds
    double elapsed_seconds = (double)(end - start) / CLOCKS_PER_SEC;

    std::vector<std::string> result_string;
    for (const auto &atom : result.atoms()) {
        result_string.push_back(atom->getSymbol());
    }


    cout << "[";
    for ( int idx = 0; idx < result_string.size(); idx++ ){
        if(idx == result_string.size()-1 ){
            cout <<"'"<<result_string.at(idx)<<"']"<<endl;
        }
        else cout <<"'"<<result_string.at(idx)<<"', ";
    }
    cout<<"done\n\n";

     // Print the elapsed time in seconds
    std::cout << "\nElapsed time: " << elapsed_seconds << " seconds" << std::endl;

    return 0;
}
